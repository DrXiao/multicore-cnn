#include <stdio.h>
#include <stdlib.h>
#include <stdint.h>
#include <time.h>
#include <hip/hip_runtime.h>
#include "util.h"

int main(int argc, char **argv) {

    if (argc < 2) {
        printf("Utility:\n\tmain [image.bin]\n");
        exit(0);
    }
    printf("==== Doing Convolution for all files ====\n");
    printf("Thread Block %d, Threads %d\n", THREAD_BLOCK, THREADS);
    for (int img_idx = 1; img_idx < argc; img_idx++)
        cnn(argv[img_idx]);
    printf("==== Ending Convolution for all files ====\n");
    return 0;
}
