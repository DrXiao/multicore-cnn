#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include "util.h"
#include "cnn_cpu.h"
#include "cnn_gpu.h"
#define __CPU__ "CPU"
#define __GPU__ "GPU"
#define __SOBEL__ "sobel"
#define __GAUSSIAN__ "gaussian"

void img_dump(int origin_len, unsigned char *img, int img_len, const char *compute_type,
              const char *kernel) {
    FILE *new_file;
    char filename[64];
    sprintf(filename, "%s_%s_%d.bin", compute_type, kernel, origin_len);
    new_file = fopen(filename, "wb");
    for (int i = 0; i < (img_len * img_len); i++) {
        fprintf(new_file, "%c", img[i]);
    }
    fclose(new_file);
}

void cnn(char *argv) {
    int image_len = 0;
    sscanf(argv, "img/%d.bin", &image_len);

    FILE *image = fopen(argv, "rb");
    printf("== %s ==\n", argv);
    if (image == NULL) {
        printf("%s\n", argv);
        printf("Error! No image!\n");
        return;
    }

    unsigned char *image_arr = NULL;
    unsigned char *cuda_copy_img = NULL;
    image_arr = (unsigned char *)calloc(image_len * image_len, sizeof(unsigned char));

    int idx = 0;

    while (fscanf(image, "%c", image_arr + idx) != EOF) {
        idx++;
    }
    hipMalloc((void **)&cuda_copy_img, image_len * image_len * sizeof(unsigned char));
    hipMemcpy((void *)cuda_copy_img, (void *)image_arr,
               image_len * image_len * sizeof(unsigned char), hipMemcpyHostToDevice);

               unsigned char *cpu_new_img = NULL, *gpu_new_img = NULL;
               unsigned char *return_img = NULL;
    int new_len = 0;
    float gpu_elapsed_time = 0;
    hipEvent_t start, end;

    /* Sobel kernel by CPU */
    printf("%20s - ", "Sobel Kernel CPU");
    cpu_new_img =
        convolution_by_CPU(image_arr, image_len, sobel_compute, &new_len);
    img_dump(image_len, cpu_new_img, new_len, __CPU__, __SOBEL__);
    /* End Sobel kernel by CPU */
    /* Sobel kernel by GPU */
    gpu_new_img = (unsigned char *)malloc(new_len * new_len * sizeof(unsigned char));
    hipMalloc((void **)&return_img, new_len * new_len * sizeof(unsigned char));
    hipEventCreate(&start);
    hipEventCreate(&end);
    hipEventRecord(start, 0);
    convolution_by_sobel_GPU<<<THREAD_BLOCK, THREADS>>>(
        cuda_copy_img, image_len, return_img);
    hipEventRecord(end, 0);
    hipEventSynchronize(end);
    hipEventElapsedTime(&gpu_elapsed_time, start, end);
    hipMemcpy((void *)gpu_new_img, (void *)return_img,
               new_len * new_len * sizeof(unsigned char), hipMemcpyDeviceToHost);

    printf("%20s - ", "Sobel Kernel GPU");
    printf("%d -> %d : %3.20f s\n", image_len, new_len,
           (float)(gpu_elapsed_time) / (float)1000);
    img_dump(image_len, gpu_new_img, new_len, __GPU__, __SOBEL__);
    /* End Sobel kernel by GPU */

    hipFree(return_img);
    free(cpu_new_img);
    free(gpu_new_img);

    /* Gaussian kernel by CPU */
    printf("%20s - ", "Gaussian Kernel CPU");
    cpu_new_img =
        convolution_by_CPU(image_arr, image_len, gaussian_compute, &new_len);
    img_dump(image_len, cpu_new_img, new_len, __CPU__, __GAUSSIAN__);
    /* End Gaussian kernel by CPU *
    /* Gaussian kernel by GPU */
    gpu_new_img = (unsigned char *)malloc(new_len * new_len * sizeof(unsigned char));
    hipMalloc((void **)&return_img, new_len * new_len * sizeof(unsigned char));
    hipEventCreate(&start);
    hipEventCreate(&end);
    hipEventRecord(start, 0);
    convolution_by_gaussian_GPU<<<THREAD_BLOCK, THREADS>>>(
        cuda_copy_img, image_len, return_img);
    hipEventRecord(end, 0);
    hipEventSynchronize(end);
    hipEventElapsedTime(&gpu_elapsed_time, start, end);
    hipMemcpy((void *)gpu_new_img, (void *)return_img,
               new_len * new_len * sizeof(unsigned char), hipMemcpyDeviceToHost);

    printf("%20s - ", "Gaussian Kernel GPU");
    printf("%d -> %d : %3.20f s\n", image_len, new_len,
           (float)(gpu_elapsed_time) / (float)1000);
    img_dump(image_len, gpu_new_img, new_len, __GPU__, __GAUSSIAN__);
    /* End Gaussian kernel by GPU */
    hipFree(return_img);
    free(cpu_new_img);
    free(gpu_new_img);

    hipFree(cuda_copy_img);
    free(image_arr);
    fclose(image);
}
