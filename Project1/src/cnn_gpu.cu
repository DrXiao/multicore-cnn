#include "hip/hip_runtime.h"
#include <stdio.h>
#include <math.h>
#include <hip/hip_runtime.h>
#include "cnn_gpu.h"
#include "util.h"
#define START_IDX blockDim.x *blockIdx.x + threadIdx.x

__device__ unsigned char sobel_gpu_compute(unsigned char *img, int img_len) {
    static char sobel_gpu[9] = {1, 0, -1, 2, 0, -2, 1, 0, -1};
    int result;
    result = 0;
    for (int row = 0; row < 3; row++) {
        for (int col = 0; col < 3; col++) {
            result +=
                sobel_gpu[row * 3 + col] * img[(row - 1) * img_len + (col - 1)];
        }
    }
    if (result > 255)
        result = 255;
    else if (result < 0)
        result = 0;
    return (unsigned char)result;
}

__device__ unsigned char gaussian_gpu_compute(unsigned char *img, int img_len) {

    static unsigned char gaussian_gpu[25] = {1,  4, 6,  4,  1,  4, 16, 24, 16,
                                             4,  6, 24, 36, 24, 6, 4,  16, 24,
                                             16, 4, 1,  4,  6,  4, 1};
    int result;
    result = 0;
    for (int row = 0; row < 5; row++) {
        for (int col = 0; col < 5; col++) {
            result += gaussian_gpu[row * 5 + col] *
                      img[(row - 2) * img_len + (col - 2)];
        }
    }
    result = result >> 8;
    if (result > 255)
        result = 255;
    else if (result < 0)
        result = 0;
    return (unsigned char)result;
}

__global__ void convolution_by_sobel_GPU(unsigned char *image, int img_len,
                                         unsigned char *return_img) {
    // TODO

    int new_len;
    int offset;

    new_len = img_len - 2;
    offset = 1;

    // Doing Convolution
    /*
        for (int new_row = START_IDX; new_row < new_len; new_row += THREADS)
       { for (int new_col = 0; new_col < new_len; new_col++) {
                // res = gaussian_gpu_compute(origin_img_ptr, img_len);
                return_img[new_row * new_len + new_col] = sobel_gpu_compute(
                    image + (new_row + offset) * img_len + (new_col + offset),
                    img_len);
            }
        }
    */

    /*
        ex :    4 Blocks, 2 Threads per block.
                original image : 17 * 17

                new image : 15 * 15 = 225

        total_threads = 4 * 2 = 8
        jump = (15 * 15) / 8 = 28       // (15 * 15) % 8 = 1
        jump += 1      ==> 29

        1st thread
            idx = [0, 29)
        2st thread
            idx = [29, 58)
        .
        .
        .
        8th thread
            idx = [203, 225)            // min((8 - 1) * 29, 15 * 15) =>
       min(232, 225)
    */

    int total_threads = THREADS * THREAD_BLOCK;
    int jump = (new_len * new_len) / total_threads;
    jump += (new_len * new_len) % total_threads == 0 ? 0 : 1;
    int startIdx = START_IDX;
    for (int idx = startIdx * jump;
         idx < min((startIdx + 1) * jump, new_len * new_len); idx++) {
        return_img[idx] =
            sobel_gpu_compute(image + (idx / new_len + offset) * img_len +
                                  (idx % new_len + offset),
                              img_len);
    }
}

__global__ void convolution_by_gaussian_GPU(unsigned char *image, int img_len,
                                            unsigned char *return_img) {
    // TODO

    int new_len;
    int offset;

    new_len = img_len - 4;
    offset = 2;

    // Doing Convolution
    /*
        for (int new_row = START_IDX; new_row < new_len; new_row += THREADS)
       { for (int new_col = 0; new_col < new_len; new_col++) {
                // res = gaussian_gpu_compute(origin_img_ptr, img_len);
                return_img[new_row * new_len + new_col] = gaussian_gpu_compute(
                    image + (new_row + offset) * img_len + (new_col + offset),
                    img_len);
            }
        }
    */

    int total_threads = THREADS * THREAD_BLOCK;
    int jump = (new_len * new_len) / total_threads;
    jump += (new_len * new_len) % total_threads == 0 ? 0 : 1;
    int startIdx = START_IDX;
    for (int idx = startIdx * jump;
         idx < min((startIdx + 1) * jump, new_len * new_len); idx++) {
        return_img[idx] =
            gaussian_gpu_compute(image + (idx / new_len + offset) * img_len +
                                     (idx % new_len + offset),
                                 img_len);
    }
}