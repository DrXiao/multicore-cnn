#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <time.h>
#include "util.h"
#include "cnn_cpu.h"

unsigned char sobel_compute(unsigned char *img, int img_len) {
    static char sobel[9] = {1, 0, -1, 2, 0, -2, 1, 0, -1};
    int result;
    result = 0;
    for (int row = 0; row < 3; row++) {
        for (int col = 0; col < 3; col++) {
            result +=
                sobel[row * 3 + col] * img[(row - 1) * img_len + (col - 1)];
        }
    }
    if (result > 255)
        result = 255;
    else if (result < 0)
        result = 0;
    return (unsigned char)result;
}

unsigned char gaussian_compute(unsigned char *img, int img_len) {

    static unsigned char gaussian_blur[25] = {1,  4, 6,  4,  1,  4, 16, 24, 16,
                                              4,  6, 24, 36, 24, 6, 4,  16, 24,
                                              16, 4, 1,  4,  6,  4, 1};
    int result;
    result = 0;
    for (int row = 0; row < 5; row++) {
        for (int col = 0; col < 5; col++) {
            result += gaussian_blur[row * 5 + col] *
                      img[(row - 2) * img_len + (col - 2)];
        }
    }
    // printf("result : %d\n", result);
    result = result >> 8;
    if (result > 255)
        result = 255;
    else if (result < 0)
        result = 0;
    return (unsigned char)result;
}

unsigned char *convolution_by_CPU(unsigned char *image, int img_len,
                                  unsigned char (*kernel)(unsigned char *, int),
                                  int *return_len) {
    unsigned char *new_img;
    int new_len;
    int offset;
    clock_t start, end;
    start = clock();
    if (kernel == sobel_compute) {
        new_len = img_len - (3 - 1);
        offset = 3 >> 1;
    }
    else {
        new_len = img_len - (5 - 1);
        offset = 5 >> 1;
    }
    new_img =
        (unsigned char *)malloc(new_len * new_len * sizeof(unsigned char));

    // Doing Convolution

    for (int new_row = 0; new_row < new_len; new_row++) {
        for (int new_col = 0; new_col < new_len; new_col++) {
            new_img[new_row * new_len + new_col] = kernel(
                image + (new_row + offset) * img_len + (new_col + offset),
                img_len);
        }
    }
    *return_len = new_len;
    end = clock();
    printf("%d -> %d : %3.20f s\n", img_len, new_len,
           (float)(end - start) / (float)CLOCKS_PER_SEC);
    return new_img;
}
